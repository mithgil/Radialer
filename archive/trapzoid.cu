
#include <hip/hip_runtime.h>
#include <iostream>

// CUDA kernel for Trapezoidal rule integration
__global__ void trapezoidalIntegration(float *x, float *y, float *z, int n, float *result) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < n) {
        result[tid] = 0.5 * (z[tid] + z[tid + 1]) * (x[tid + 1] - x[tid]) * (y[tid + 1] - y[tid]);
    }
}

int main() {
    const int n = 1000; // Number of intervals
    const int size = n + 1; // Number of points

    // Host arrays
    float *x = new float[size];
    float *y = new float[size];
    float *z = new float[size];

    // Initialize x, y, and z arrays on the host

    // Device arrays
    float *d_x, *d_y, *d_z, *d_result;
    hipMalloc(&d_x, size * sizeof(float));
    hipMalloc(&d_y, size * sizeof(float));
    hipMalloc(&d_z, size * sizeof(float));
    hipMalloc(&d_result, n * sizeof(float));

    // Copy input data from host to device
    hipMemcpy(d_x, x, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, z, size * sizeof(float), hipMemcpyHostToDevice);

    // Configure grid and block dimensions
    int block_size = 256;
    int num_blocks = (n + block_size - 1) / block_size;

    // Launch CUDA kernel
    trapezoidalIntegration<<<num_blocks, block_size>>>(d_x, d_y, d_z, n, d_result);

    // Copy result from device to host
    float *result = new float[n];
    hipMemcpy(result, d_result, n * sizeof(float), hipMemcpyDeviceToHost);

    // Calculate the total integral result
    float total_integral = 0.0;
    for (int i = 0; i < n; ++i) {
        total_integral += result[i];
    }

    // Clean up
    delete[] x;
    delete[] y;
    delete[] z;
    delete[] result;
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_z);
    hipFree(d_result);

    std::cout << "Total Integral: " << total_integral << std::endl;

    return 0;
}
